#include "hip/hip_runtime.h"
/*
 * Copyright 2025 Nvidia Corporation
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *     http://www.apache.org/licenses/LICENSE-2.0
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
**/

#include <cupqc.hpp>
#include <stdexcept>
#include <oqs/oqsconfig.h>

using namespace cupqc;

// Checks the return value from a CUDA API function
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { failure = true; goto cleanup; }

template<class MLKEM_Keygen>
__global__ void keygen_kernel(uint8_t *pk, uint8_t *sk, uint8_t *workspace, uint8_t *randombytes) {
    __shared__ uint8_t smem_ptr[MLKEM_Keygen::shared_memory_size];
    MLKEM_Keygen().execute(pk, sk, randombytes, workspace, smem_ptr);
}

template<class MLKEM_Base>
int keypair(uint8_t *pk, uint8_t *sk) {
    using MLKEM_Keygen = decltype(MLKEM_Base() + Function<function::Keygen>());

    bool failure = false;
    uint8_t *workspace = nullptr, *randombytes=nullptr;
    uint8_t *d_pk = nullptr, *d_sk = nullptr;

    // Allocate device workspaces
    try {
        workspace   = make_workspace<MLKEM_Keygen>(1);
        randombytes = get_entropy<MLKEM_Keygen>(1);
    } catch (const std::runtime_error& ex) {
        failure = true;
        goto cleanup;
    }
    CUDA_CHECK(hipMalloc((void**)&d_pk, MLKEM_Keygen::public_key_size));
    CUDA_CHECK(hipMalloc((void**)&d_sk, MLKEM_Keygen::secret_key_size));

    // Run routine
    keygen_kernel<MLKEM_Keygen><<<1, MLKEM_Keygen::BlockDim>>>(d_pk, d_sk, workspace, randombytes);

    // Copy data back to the host
    CUDA_CHECK(hipMemcpy(pk, d_pk, MLKEM_Keygen::public_key_size, hipMemcpyDefault));
    CUDA_CHECK(hipMemcpy(sk, d_sk, MLKEM_Keygen::secret_key_size, hipMemcpyDefault));

cleanup:
    // Free device memory
    if (d_pk != nullptr) hipFree(d_pk);
    if (d_sk != nullptr) hipFree(d_sk);
    if (workspace != nullptr) destroy_workspace(workspace);
    if (randombytes != nullptr) release_entropy(randombytes);

    return failure ? -1 : 0;
}

template<class MLKEM_Encaps>
__global__ void encaps_kernel(uint8_t *ct, uint8_t *ss, const uint8_t *pk, uint8_t *workspace, uint8_t *randombytes) {
    __shared__ uint8_t smem_ptr[MLKEM_Encaps::shared_memory_size];
    MLKEM_Encaps().execute(ct, ss, pk, randombytes, workspace, smem_ptr);
}

template<class MLKEM_Base>
int encaps(uint8_t *ct, uint8_t *ss, const uint8_t *pk) {
    using MLKEM_Encaps = decltype(MLKEM_Base() + Function<function::Encaps>());

    bool failure = false;
    uint8_t *workspace = nullptr, *randombytes=nullptr;
    uint8_t *d_ct = nullptr, *d_ss = nullptr, *d_pk = nullptr;

    // Allocate device workspaces
    try {
        workspace   = make_workspace<MLKEM_Encaps>(1);
        randombytes = get_entropy<MLKEM_Encaps>(1);
    } catch (const std::runtime_error& ex) {
        failure = true;
        goto cleanup;
    }
    CUDA_CHECK(hipMalloc((void**)&d_ct, MLKEM_Encaps::ciphertext_size));
    CUDA_CHECK(hipMalloc((void**)&d_ss, MLKEM_Encaps::shared_secret_size));
    CUDA_CHECK(hipMalloc((void**)&d_pk, MLKEM_Encaps::public_key_size));

    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_pk, pk, MLKEM_Encaps::public_key_size, hipMemcpyDefault));

    // Run routine
    encaps_kernel<MLKEM_Encaps><<<1, MLKEM_Encaps::BlockDim>>>(d_ct, d_ss, d_pk, workspace, randombytes);

    // Copy data back to the host
    CUDA_CHECK(hipMemcpy(ct, d_ct, MLKEM_Encaps::ciphertext_size, hipMemcpyDefault));
    CUDA_CHECK(hipMemcpy(ss, d_ss, MLKEM_Encaps::shared_secret_size, hipMemcpyDefault));

cleanup:
    // Free device memory
    if (d_ct != nullptr) hipFree(d_ct);
    if (d_ss != nullptr) hipFree(d_ss);
    if (d_pk != nullptr) hipFree(d_pk);
    if (workspace != nullptr) destroy_workspace(workspace);
    if (randombytes != nullptr) release_entropy(randombytes);

    return failure ? -1 : 0;
}

template<class MLKEM_Decaps>
__global__ void decaps_kernel(uint8_t *ss, const uint8_t *ct, const uint8_t *sk, uint8_t *workspace) {
    __shared__ uint8_t smem_ptr[MLKEM_Decaps::shared_memory_size];
    MLKEM_Decaps().execute(ss, ct, sk, workspace, smem_ptr);
}

template<class MLKEM_Base>
int decaps(uint8_t *ss, const uint8_t *ct, const uint8_t *sk) {
    using MLKEM_Decaps = decltype(MLKEM_Base() + Function<function::Decaps>());

    bool failure = false;
    uint8_t *workspace = nullptr;
    uint8_t *d_ct = nullptr, *d_ss = nullptr, *d_sk = nullptr;

    // Allocate device workspaces
    try {
        workspace = make_workspace<MLKEM_Decaps>(1);
    } catch (const std::runtime_error& ex) {
        failure = true;
        goto cleanup;
    }
    CUDA_CHECK(hipMalloc((void**)&d_ct, MLKEM_Decaps::ciphertext_size));
    CUDA_CHECK(hipMalloc((void**)&d_ss, MLKEM_Decaps::shared_secret_size));
    CUDA_CHECK(hipMalloc((void**)&d_sk, MLKEM_Decaps::secret_key_size));

    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_sk, sk, MLKEM_Decaps::secret_key_size, hipMemcpyDefault));
    CUDA_CHECK(hipMemcpy(d_ct, ct, MLKEM_Decaps::ciphertext_size, hipMemcpyDefault));

    // Run routine
    decaps_kernel<MLKEM_Decaps><<<1, MLKEM_Decaps::BlockDim>>>(d_ss, d_ct, d_sk, workspace);

    // Copy data back to the host
    CUDA_CHECK(hipMemcpy(ss, d_ss, MLKEM_Decaps::shared_secret_size, hipMemcpyDefault));

cleanup:
    // Free device memory
    if (d_ct != nullptr) hipFree(d_ct);
    if (d_ss != nullptr) hipFree(d_ss);
    if (d_sk != nullptr) hipFree(d_sk);
    if (workspace != nullptr) destroy_workspace(workspace);

    return failure ? -1 : 0;
}

extern "C" {
    using KEM_512  = decltype(ML_KEM_512()  + Block());

#if defined(OQS_ENABLE_KEM_ml_kem_512_cuda)
    int cupqc_ml_kem_512_keypair(uint8_t *pk, uint8_t *sk) {
        return keypair<KEM_512>(pk, sk);
    }
    int cupqc_ml_kem_512_enc(uint8_t *ct, uint8_t *ss, const uint8_t *pk) {
        return encaps<KEM_512>(ct, ss, pk);
    }
    int cupqc_ml_kem_512_dec(uint8_t *ss, const uint8_t *ct, const uint8_t *sk) {
        return decaps<KEM_512>(ss, ct, sk);
    }
#endif
}
